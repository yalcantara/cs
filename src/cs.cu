#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : cs.cu
 Author      : Yaison Alcantara Alcantara
 Version     :
 Copyright   : 
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <cs/core/Exception.h>
#include <cs/core/lang.h>
#include <cs/math/CpuMatrix.h>
#include <cs/math/GpuMatrix.h>
#include <cs/math/GpuVector.h>
#include <cs/math/math.h>
#include <cs/nn/Affine.h>
#include <stddef.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <time.h>

using namespace std;
using namespace cs::core;
using namespace cs::math;
using namespace cs::nn;

void performance() {
	
	size_t d = 1000;
	GpuMatrix a = randn(d, d);
	GpuMatrix b = randn(d, d);
	
	CpuMatrix ac = randn(d, d);
	CpuMatrix bc = randn(d, d);
	
	for (int i = 0; i < 5; i++) {
		time_t now = clock();
		auto c = a.dot(b);
		auto cpu = c.cpu();
		double took = clock() - now;
		double millis = took / CLOCKS_PER_SEC * 1000.0;
		printf("millis: %8d\n", (int) millis);
	}
	
	println();
	for (int i = 0; i < 5; i++) {
		time_t now = clock();
		auto cc = ac.dot(bc);
		double took = clock() - now;
		double millis = took / CLOCKS_PER_SEC * 1000.0;
		printf("millis: %8d\n", (int) millis);
	}
}

void test1() {
	GpuMatrix a = { { 1, 2, 3 }, { 4, 5, 6 } };
	GpuMatrix b = { { 1, 2 }, { 3, 4 }, { 5, 6 } };
	
	auto scal = 2 * a;
	scal.print();
	
	auto d = a.dot(b);
	d.print();
	GpuVector v = { 1, 2 };
	auto c = a.affine(b, v);
	
	c.print();
}

int main(void) {
	try {
		srand(time(NULL));
		
		Affine* l = new Affine();
		l->use_gpu(false);
		
		CpuMatrix x = { { 1, 2 }, { 3, 4 } };
		CpuMatrix y = { { 0 }, { 1 } };
		
		Matrix& base = x;
		base.print();
		
		l->set_dim(x.n, 1);
		l->init();
		
		
		println("klk");
	} catch (Exception& ex) {
		println("Exception thrown");
		println(ex.what());
	}
	return 0;
}

