#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : cs.cu
 Author      : Yaison Alcantara Alcantara
 Version     :
 Copyright   : 
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <cs/core/lang.h>
#include <cs/math/CpuMatrix.h>
#include <cs/math/GpuMatrix.h>
#include <cs/math/GpuVector.h>
#include <cs/math/math.h>
#include <stddef.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <time.h>
#include <cs/nn/Layer.h>
#include <cs/nn/Affine.h>

using namespace std;
using namespace cs::core;
using namespace cs::math;
using namespace cs::nn;

void performance() {
	
	size_t d = 1000;
	GpuMatrix a = randn(d, d);
	GpuMatrix b = randn(d, d);
	
	CpuMatrix ac = randn(d, d);
	CpuMatrix bc = randn(d, d);
	
	for (int i = 0; i < 5; i++) {
		time_t now = clock();
		auto c = a.dot(b);
		auto cpu = c.cpu();
		double took = clock() - now;
		double millis = took / CLOCKS_PER_SEC * 1000.0;
		printf("millis: %8d\n", (int) millis);
	}
	
	println();
	for (int i = 0; i < 5; i++) {
		time_t now = clock();
		auto cc = ac.dot(bc);
		double took = clock() - now;
		double millis = took / CLOCKS_PER_SEC * 1000.0;
		printf("millis: %8d\n", (int) millis);
	}
}

void test1() {
	GpuMatrix a = { { 1, 2, 3 }, { 4, 5, 6 } };
	GpuMatrix b = { { 1, 2 }, { 3, 4 }, { 5, 6 } };
	
	auto scal = 2 * a;
	scal.print();
	
	auto d = a.dot(b);
	d.print();
	GpuVector v = { 1, 2 };
	auto c = a.affine(b, v);
	
	c.print();
}

int main(void) {
	srand(time(NULL));
	
	Layer* l = new Affine();

	
	GpuMatrix x = { { 1, 2 }, { 3, 4 } };
	
	Matrix& fx = l->foward(x);
	
	fx.print();
	
	println("klk");
	return 0;
}

