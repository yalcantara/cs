#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : cs.cu
 Author      : Yaison Alcantara Alcantara
 Version     :
 Copyright   : 
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <cs/core/Exception.h>
#include <cs/core/lang.h>
#include <cs/core/utils.h>
#include <cs/data/Grid.h>
#include <cs/data/GridInfo.h>
#include <cs/gpu/gpu.h>
#include <cs/math/CpuMatrix.h>
#include <cs/math/CpuVector.h>
#include <cs/math/GpuMatrix.h>
#include <cs/math/GpuVector.h>
#include <cs/math/math.h>
#include <cs/nn/Affine.h>
#include <cs/nn/errors.h>
#include <cs/nn/MinSquare.h>
#include <cs/nn/Network.h>
#include <cs/nn/Sigmoid.h>
#include <stddef.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <time.h>
#include <string>

using namespace std;
using namespace cs::core;
using namespace cs::math;
using namespace cs::nn;
using namespace cs::gpu;
using namespace cs::data;

void memtest() {
	//test that the memory is ok by doing some memory allocation/dealocation etc...
	
	float* test = (float*) malloc(sizeof(float) * 5);
	free(test);
}

void hit(CpuMatrix& h, CpuMatrix& y) {
	
	size_t total = y.m;
	size_t classes = y.n;
	size_t good = 0;
	
	for (size_t i = 0; i < total; i++) {
		for (size_t j = 0; j < classes; j++) {
			float val = y.get(i, j);
			float ans = h.get(i, j);
			
			if (val == 1) {
				if (ans >= 0.5) {
					good++;
				}
			} else {
				if (ans < 0.5) {
					good++;
				}
			}
		}
	}
	
	println("======================================================");
	printf("total: %6d\n", (int) total);
	printf("good : %6d\n", (int) good);
	printf("perc : %6.0f%\n", (float) (good * 100.0 / (total * classes)));
	println("======================================================");
}

void performance() {
	
	size_t d = 1000;
	GpuMatrix a = randn(d, d);
	GpuMatrix b = randn(d, d);
	
	CpuMatrix ac = randn(d, d);
	CpuMatrix bc = randn(d, d);
	
	for (int i = 0; i < 5; i++) {
		time_t now = clock();
		auto c = a.dot(b);
		auto cpu = c.cpu();
		double took = clock() - now;
		double millis = took / CLOCKS_PER_SEC * 1000.0;
		printf("millis: %8d\n", (int) millis);
	}
	
	println();
	for (int i = 0; i < 5; i++) {
		time_t now = clock();
		auto cc = ac.dot(bc);
		double took = clock() - now;
		double millis = took / CLOCKS_PER_SEC * 1000.0;
		printf("millis: %8d\n", (int) millis);
	}
}

void test1() {
	GpuMatrix a = { { 1, 2, 3 }, { 4, 5, 6 } };
	GpuMatrix b = { { 1, 2 }, { 3, 4 }, { 5, 6 } };
	
	auto scal = 2 * a;
	scal.print();
	
	auto d = a.dot(b);
	d.print();
	GpuVector v = { 1, 2 };
	auto c = a.affine(b, v);
	
	c.print();
}

void test2() {
	try {
		srand(time(NULL));
		
		Affine f = Affine();
		f.use_gpu(false);
		CpuMatrix x = { { 0 }, { 1 } };
		
		CpuMatrix y = { { 1 }, { 0 } };
		
		println("X:");
		x.print();
		
		println("Y");
		y.print();
		
		f.set_dim(x.n, y.n);
		f.init();
		CpuMatrix w = { { 0 } };
		CpuVector b = { 2 };
		f.set_weights(w);
		f.set_bias(b);
		
		float j;
		float alpha = 0.1;
		
		//f.print();
		int iter = 100;
		for (int i = 0; i <= iter; i++) {
			//f.print();
			Matrix& h = f.foward(x);
			if (iter < 10 || i % (iter / 10) == 0) {
				j = min_square_error(h, y);
				printf("iter: %6d,  j: %12.8f\n", i, j);
			}
			
			CpuMatrix dg = cpu_cast(h) - y;
			
			f.backward(dg);
			f.update(alpha);
			
		}
		
		println("ended");
	} catch (Exception& ex) {
		println("Exception thrown");
		println(ex.what());
	}
}

void gpu_test() {
	try {
		srand(time(NULL));
		
		Affine f = Affine();
		f.use_gpu(true);
		GpuMatrix x = { { 0, 0 }, { 0, 1 } };
		
		GpuMatrix y = { { 1 }, { 0 } };
		
		println("X:");
		x.print();
		
		println("Y");
		y.print();
		
		f.set_dim(x.n, y.n);
		f.init();
		println("Affine initialized");
		GpuMatrix w = { { -1 }, { 5 } };
		
		GpuVector b = { 2 };
		f.set_weights(w);
		println("Weights set");
		f.set_bias(b);
		println("Bias set");
		float j;
		float alpha = 0.1;
		
		f.print();
		println("About to train");
		int iter = 1000;
		for (int i = 0; i <= iter; i++) {
			//println("===================================");
			//f.print();
			
			Matrix& h = f.foward(x);
			
			if (iter <= 10 || i % (iter / 10) == 0) {
				j = min_square_error(h, y);
				//println("=======================================");
				printf("iter: %6d  J: %12.8f", i, j);
				println();
			}
			
			GpuMatrix dg = gpu_cast(h) - y;
			
			f.backward(dg);
			
			f.update(alpha);
		}
		
		println();
		Matrix& h = f.foward(x);
		
		f.print();
		h.print();
		println("ended");
	} catch (Exception& ex) {
		println("Exception thrown");
		println(ex.what());
	}
}

void trans_test1() {
	
	println("===================================================");
	println("A^T x B  case 1");
	//continue here: test sigmoid
	GpuMatrix a = { { 1, 2 }, { 3, 4 } };
	GpuMatrix b = { { 1, 0, 0 }, { 3, 1, 1 } };
	
	GpuMatrix c = GpuMatrix(a.n, b.n);
	
	size_t m = a.m;
	size_t n = a.n;
	size_t p = b.n;
	float* A = a.ptr();
	float* B = b.ptr();
	float* C = c.ptr();
	
	gpu_dot(A, true, B, C, m, n, p);
	
	println("The ans should be:");
	GpuMatrix ans = { { 10, 3, 3 }, { 14, 4, 4 } };
	ans.print();
	
	println("Got:");
	c.print();
}

void trans_test2() {
	println("===================================================");
	println("A^T x B  case 2");
	//continue here: test sigmoid
	GpuMatrix a = { { 1, 2, 0 }, { 1, 0, 1 } };
	GpuMatrix b = { { 1, 2, 3, 0 }, { 2, 0, 3, 5 } };
	
	GpuMatrix c = GpuMatrix(a.n, b.n);
	
	size_t m = a.m;
	size_t n = a.n;
	size_t p = b.n;
	float* A = a.ptr();
	float* B = b.ptr();
	float* C = c.ptr();
	
	gpu_dot(A, true, B, C, m, n, p);
	
	println("The ans should be:");
	GpuMatrix ans = { { 3, 2, 6, 5 }, { 2, 4, 6, 0 }, { 2, 0, 3, 5 } };
	ans.print();
	
	println("Got:");
	c.print();
}

void trans_test3() {
	println("===================================================");
	println("A x B^T  case 1");
	//continue here: test sigmoid
	GpuMatrix a = { { 1, 2 }, { 3, 4 } };
	GpuMatrix b = { { 1, 0 }, { 0, 3 }, { 1, 1 } };
	
	GpuMatrix c = GpuMatrix(a.m, b.m);
	
	size_t m = a.m;
	size_t n = a.n;
	size_t o = b.m;
	size_t p = b.n;
	float* A = a.ptr();
	float* B = b.ptr();
	float* C = c.ptr();
	
	b.print();
	gpu_dot(A, B, true, C, m, n, o, p);
	
	println("The ans should be:");
	GpuMatrix ans = { { 1, 6, 3 }, { 3, 12, 7 } };
	ans.print();
	
	println("Got:");
	c.print();
}

void trans_test4() {
	println("===================================================");
	println("A x B^T  case 2");
	//continue here: test sigmoid
	GpuMatrix a = { { 1, 2 }, { 1, 0 }, { 4, 5 } };
	GpuMatrix b = { { 1, 2 }, { 3, 0 } };
	
	GpuMatrix c = GpuMatrix(a.m, b.m);
	
	size_t m = a.m;
	size_t n = a.n;
	size_t o = b.m;
	size_t p = b.n;
	float* A = a.ptr();
	float* B = b.ptr();
	float* C = c.ptr();
	
	gpu_dot(A, B, true, C, m, n, o, p);
	
	println("The ans should be:");
	GpuMatrix ans = { { 5, 3 }, { 1, 3 }, { 14, 12 } };
	ans.print();
	
	println("Got:");
	c.print();
}

void sigmoid_test() {
	
	GpuMatrix a = { { 1, -1 }, { -16, 16 }, { -100, 100 }, { 0, 0 } };
	
	GpuMatrix b = GpuMatrix(a.m, a.n);
	
	a.print();
	Sigmoid s = Sigmoid();
	s.use_gpu(true);
	s.set_dim(a.n);
	
	Matrix& ans = s.foward(a);
	
	ans.print();
}

void sigmoid_test2() {
	try {
		srand(time(NULL));
		
		GpuMatrix x = { { 0, 0 }, { 0, 1 }, { 1, 0 }, { 1, 1 } };
		GpuMatrix y = { { 1 }, { 0 }, { 0 }, { 1 } };
		
		println("X:");
		x.print();
		
		println("Y");
		y.print();
		
		Affine f = Affine();
		f.use_gpu(true);
		f.set_dim(x.n, y.n);
		f.init();
		
		Sigmoid s = Sigmoid();
		s.use_gpu(true);
		s.set_dim(f.out_dim());
		s.init();
		
		println("Bias set");
		float j;
		float alpha = 0.1;
		
		f.print();
		println("About to train");
		int iter = 10000;
		for (int i = 0; i <= iter; i++) {
			//println("===================================");
			//f.print();
			
			Matrix& h1 = f.foward(x);
			Matrix& h2 = s.foward(h1);
			
			if (iter <= 10 || i % (iter / 10) == 0) {
				j = min_square_error(h2, y);
				//println("=======================================");
				printf("iter: %6d  J: %12.8f", i, j);
				println();
			}
			
			GpuMatrix dg = gpu_cast(h2) - y;
			Matrix& b = s.backward(dg);
			f.backward(b);
			
			f.update(alpha);
		}
		
		println();
		Matrix& h1 = f.foward(x);
		Matrix& h2 = s.foward(h1);
		println("Ans:");
		h2.print();
		println("ended");
	} catch (Exception& ex) {
		println("Exception thrown");
		println(ex.what());
	}
}

void sigmoid_test3() {
	try {
		srand(time(NULL));
		
		CpuMatrix x = { { 0, 0 }, { 0, 1 }, { 1, 0 }, { 1, 1 } };
		CpuMatrix y = { { 1 }, { 0 }, { 0 }, { 1 } };
		
		println("X:");
		x.print();
		
		println("Y");
		y.print();
		
		bool gpu = false;
		
		Affine f1 = Affine();
		f1.use_gpu(gpu);
		f1.set_dim(x.n, x.n);
		f1.init();
		
		Sigmoid s1 = Sigmoid();
		s1.use_gpu(gpu);
		s1.set_dim(f1.out_dim());
		s1.init();
		
		Affine f2 = Affine();
		f2.use_gpu(gpu);
		f2.set_dim(x.n, y.n);
		f2.init();
		
		Sigmoid s2 = Sigmoid();
		s2.use_gpu(gpu);
		s2.set_dim(f2.out_dim());
		s2.init();
		
		println("Bias set");
		float j;
		float alpha = 0.1;
		
		println("About to train");
		int iter = 100000;
		for (int i = 0; i <= iter; i++) {
			//println("===================================");
			//f.print();
			
			Matrix& h1 = f1.foward(x);
			Matrix& h2 = s1.foward(h1);
			Matrix& h3 = f2.foward(h2);
			Matrix& h4 = s2.foward(h3);
			
			if (iter <= 10 || i % (iter / 10) == 0) {
				j = min_square_error(h4, y);
				//println("=======================================");
				printf("iter: %8d  J: %12.8f", i, j);
				println();
			}
			
			CpuMatrix dg = cpu_cast(h4) - y;
			Matrix& b1 = s2.backward(dg);
			Matrix& b2 = f2.backward(b1);
			Matrix& b3 = s1.backward(b2);
			f1.backward(b3);
			
			f2.update(alpha);
			f1.update(alpha);
		}
		
		println();
		
		Matrix& h1 = f1.foward(x);
		Matrix& h2 = s1.foward(h1);
		Matrix& h3 = f2.foward(h2);
		Matrix& h4 = s2.foward(h3);
		
		println("Ans:");
		h4.print();
		println("ended");
	} catch (Exception& ex) {
		println("Exception thrown");
		println(ex.what());
	}
}

void iris_test_gpu() {
	
	string data = ffull("files/iris.data");
	
	GridInfo info = GridInfo(4);
	
	Grid g = Grid(data);
	//g.shuffle();
	
	GpuMatrix x = g.toMatrix(0, 4, true);
	
	GpuMatrix y = g.toMatrix(4, 5, false);
	
	bool gpu = true;
	
	Affine f1 = Affine();
	f1.use_gpu(gpu);
	f1.set_dim(x.n, x.n);
	f1.init();
	
	Sigmoid s1 = Sigmoid();
	s1.use_gpu(gpu);
	s1.set_dim(f1.out_dim());
	s1.init();
	
	Affine f2 = Affine();
	f2.use_gpu(gpu);
	f2.set_dim(s1.out_dim(), y.n);
	f2.init();
	
	Sigmoid s2 = Sigmoid();
	s2.use_gpu(gpu);
	s2.set_dim(f2.out_dim());
	s2.init();
	
	println("About to train");
	int iter = 100000;
	float j;
	float alpha = 0.1;
	for (int i = 0; i <= iter; i++) {
		//println("===================================");
		//f.print();
		
		Matrix& h1 = f1.foward(x);
		Matrix& h2 = s1.foward(h1);
		Matrix& h3 = f2.foward(h2);
		Matrix& h4 = s2.foward(h3);
		
		if (iter <= 10 || i % (iter / 10) == 0) {
			j = min_square_error(h4, y);
			//println("=======================================");
			printf("iter: %8d  J: %12.8f", i, j);
			println();
		}
		
		GpuMatrix dg = gpu_cast(h4) - y;
		Matrix& b1 = s2.backward(dg);
		Matrix& b2 = f2.backward(b1);
		Matrix& b3 = s1.backward(b2);
		f1.backward(b3);
		
		f2.update(alpha);
		f1.update(alpha);
	}
	
	println();
	Matrix& h1 = f1.foward(x);
	Matrix& h2 = s1.foward(h1);
	Matrix& h3 = f2.foward(h2);
	Matrix& h4 = s2.foward(h3);
	
	println("Ans:");
	h4.print();
	println("ended");
}

void test_data() {
	
	string data = ffull("files/iris.data");
	
	Grid g = Grid(data);
	//g.shuffle();
	
	CpuMatrix x = g.toMatrix(0, 4, false);
	
	CpuMatrix y = g.toMatrix(4, 5, false);
	
	bool gpu = false;
	
	Affine f1 = Affine();
	f1.use_gpu(gpu);
	f1.set_dim(x.n, x.n);
	f1.init();
	
	Sigmoid s1 = Sigmoid();
	s1.use_gpu(gpu);
	s1.set_dim(f1.out_dim());
	s1.init();
	
	Affine f2 = Affine();
	f2.use_gpu(gpu);
	f2.set_dim(s1.out_dim(), y.n);
	f2.init();
	
	Sigmoid s2 = Sigmoid();
	s2.use_gpu(gpu);
	s2.set_dim(f2.out_dim());
	s2.init();
	
	MinSquare ms = MinSquare();
	ms.set_dim(s2.out_dim());
	ms.init();
	
	println("About to train");
	int iter = 100000;
	float j;
	float alpha = 0.1;
	for (int i = 0; i <= iter; i++) {
		//println("===================================");
		//f.print();
		
		Matrix& h1 = f1.foward(x);
		Matrix& h2 = s1.foward(h1);
		Matrix& h3 = f2.foward(h2);
		Matrix& h4 = s2.foward(h3);
		Matrix& h5 = ms.foward(h4);
		
		if (iter <= 10 || i % (iter / 10) == 0) {
			j = min_square_error(h4, y);
			//println("=======================================");
			printf("iter: %8d  J: %12.8f", i, j);
			println();
		}
		
		Matrix& b0 = ms.backward(y);
		Matrix& b1 = s2.backward(b0);
		Matrix& b2 = f2.backward(b1);
		Matrix& b3 = s1.backward(b2);
		f1.backward(b3);
		
		f2.update(alpha);
		f1.update(alpha);
	}
	
	println();
	Matrix& h1 = f1.foward(x);
	Matrix& h2 = s1.foward(h1);
	Matrix& h3 = f2.foward(h2);
	Matrix& h4 = s2.foward(h3);
	
	println("Ans:");
	h4.print();
	println("ended");
}

void adult_data_gpu() {
	
	string data = ffull("files/adult.data");
	
	Grid g = Grid(data);
	//g.shuffle();
	
	GpuMatrix x = g.toMatrix(0, 14, true);
	CpuMatrix yy = g.toMatrix(14, 15, false);
	
	GpuMatrix y = yy.sltcols(0, 1);
	
	bool gpu = true;
	
	Affine f1 = Affine();
	f1.use_gpu(gpu);
	f1.set_dim(x.n, x.n);
	f1.init();
	
	Sigmoid s1 = Sigmoid();
	s1.use_gpu(gpu);
	s1.set_dim(f1.out_dim());
	s1.init();
	
	Affine f2 = Affine();
	f2.use_gpu(gpu);
	f2.set_dim(s1.out_dim(), y.n);
	f2.init();
	
	Sigmoid s2 = Sigmoid();
	s2.use_gpu(gpu);
	s2.set_dim(f2.out_dim());
	s2.init();
	
	Matrix& h1 = f1.foward(x);
	Matrix& h2 = s1.foward(h1);
	Matrix& h3 = f2.foward(h2);
	Matrix& h4 = s2.foward(h3);
	
	CpuMatrix h4c = gpu_cast(h4).cpu();
	CpuMatrix yc = y.cpu();
	hit(h4c, yc);
	
	println();
	
	println("About to train");
	int iter = 10;
	float j;
	float alpha = 0.001;
	for (int i = 0; i <= iter; i++) {
		//println("===================================");
		//f.print();
		
		Matrix& h1 = f1.foward(x);
		Matrix& h2 = s1.foward(h1);
		Matrix& h3 = f2.foward(h2);
		Matrix& h4 = s2.foward(h3);
		
		if (iter <= 10 || i % (iter / 10) == 0) {
			j = min_square_error(h4, y);
			//println("=======================================");
			printf("iter: %8d  J: %12.8f", i, j);
			println();
		}
		
		GpuMatrix dg = gpu_cast(h4) - y;
		Matrix& b1 = s2.backward(dg);
		Matrix& b2 = f2.backward(b1);
		Matrix& b3 = s1.backward(b2);
		f1.backward(b3);
		
		f2.update(alpha);
		f1.update(alpha);
		
		memtest();
	}
	
	println();
	Matrix& h21 = f1.foward(x);
	Matrix& h22 = s1.foward(h21);
	Matrix& h23 = f2.foward(h22);
	Matrix& h24 = s2.foward(h23);
	
	CpuMatrix h24c = gpu_cast(h24).cpu();
	CpuMatrix y2c = y.cpu();
	hit(h24c, y2c);
	
	memtest();
}

void adult_data_cpu() {
	
	string data = ffull("files/adult.data");
	
	Grid g = Grid(data);
	//g.shuffle();
	
	CpuMatrix x = g.toMatrix(0, 14, true);
	CpuMatrix yy = g.toMatrix(14, 15, false);
	
	CpuMatrix y = yy.sltcols(0, 1);
	
	bool gpu = false;
	
	Affine f1 = Affine();
	f1.use_gpu(gpu);
	f1.set_dim(x.n, x.n);
	f1.init();
	
	Sigmoid s1 = Sigmoid();
	s1.use_gpu(gpu);
	s1.set_dim(f1.out_dim());
	s1.init();
	
	Affine f2 = Affine();
	f2.use_gpu(gpu);
	f2.set_dim(s1.out_dim(), y.n);
	f2.init();
	
	Sigmoid s2 = Sigmoid();
	s2.use_gpu(gpu);
	s2.set_dim(f2.out_dim());
	s2.init();
	
	println("About to train");
	int iter = 10;
	float j;
	float alpha = 0.001;
	
	Matrix& h1 = f1.foward(x);
	Matrix& h2 = s1.foward(h1);
	Matrix& h3 = f2.foward(h2);
	Matrix& h4 = s2.foward(h3);
	
	hit(cpu_cast(h4), y);
	println();
	
	for (int i = 0; i <= iter; i++) {
		//println("===================================");
		//f.print();
		
		Matrix& h1 = f1.foward(x);
		Matrix& h2 = s1.foward(h1);
		Matrix& h3 = f2.foward(h2);
		Matrix& h4 = s2.foward(h3);
		
		if (iter <= 10 || i % (iter / 10) == 0) {
			j = min_square_error(h4, y);
			//println("=======================================");
			printf("iter: %8d  J: %12.8f", i, j);
			println();
		}
		
		CpuMatrix dg = cpu_cast(h4) - y;
		Matrix& b1 = s2.backward(dg);
		Matrix& b2 = f2.backward(b1);
		Matrix& b3 = s1.backward(b2);
		f1.backward(b3);
		
		f2.update(alpha);
		f1.update(alpha);
		
		memtest();
	}
	
	println();
	Matrix& h21 = f1.foward(x);
	Matrix& h22 = s1.foward(h21);
	Matrix& h23 = f2.foward(h22);
	Matrix& h24 = s2.foward(h23);
	
	hit(cpu_cast(h24), y);
	
	memtest();
	
}

void networktest() {
	string data = ffull("files/iris.data");
	
	Grid g = Grid(data);
	//g.shuffle();
	
	CpuMatrix x = g.toMatrix(0, 4, true);
	CpuMatrix y = g.toMatrix(4, 5, false);
	
	Network n = Network();
	n << Affine();
	n << Sigmoid();
	n << Affine();
	n << Sigmoid();
	n << MinSquare();
	
	n.init(x, y, false);
	
	println("CPU Training");
	println("================================================");
	float j = n.min_square_error();
	printf("iter: %8d  J: %12.8f", 0, j);
	println();
	for (int i = 0; i < 10; i++) {
		n.train(10000);
		j = n.min_square_error();
		printf("iter: %8d  J: %12.8f", i, j);
		println();
		
	}
	
	println("GPU Training");
	println("================================================");
	
	GpuMatrix gx = x;
	GpuMatrix gy = y;
	n.init(gx, gy, true);
	
	j = n.min_square_error();
	printf("iter: %8d  J: %12.8f", 0, j);
	fflush(stdout);
	for (int i = 0; i < 10; i++) {
		n.train(10000);
		j = n.min_square_error();
		printf("iter: %8d  J: %12.8f", i, j);
		fflush(stdout);
		println();
	}
	
	n.forward().print();
}

void net() {
	
	string data = ffull("files/iris.data");
	
	Grid g = Grid(data);
	//g.shuffle();
	
	CpuMatrix x = g.toMatrix(0, 4, false);
	
	CpuMatrix y = g.toMatrix(4, 5, false);
	
	Network n = Network();
	
	n << Affine();
	n << Sigmoid();
	n.init(x, y, false);
	
	n.forward();
	
	println();
	memtest();
}

int main(void) {
	
	println();
	
	net();
	memtest();
	println("ok");
	
	//adult_data_cpu();
	//networktest();
	//adult_data_gpu();
	//test_data();
	
	//iris_test_gpu();
	//sigmoid_test2();
	//gpu_test();
	//test2();
	
	return 0;
}

