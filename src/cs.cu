#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : cs.cu
 Author      : Yaison Alcantara Alcantara
 Version     :
 Copyright   : 
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <cs/core/Exception.h>
#include <cs/core/lang.h>
#include <cs/math/CpuMatrix.h>
#include <cs/math/GpuMatrix.h>
#include <cs/math/GpuVector.h>
#include <cs/math/math.h>
#include <cs/nn/Affine.h>
#include <cs/nn/errors.h>
#include <stddef.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <time.h>
#include <cs/nn/gpu_layers.cuh>
#include <hip/hip_runtime.h>
#include <hipblas.h>

using namespace std;
using namespace cs::core;
using namespace cs::math;
using namespace cs::nn;

void performance() {
	
	size_t d = 1000;
	GpuMatrix a = randn(d, d);
	GpuMatrix b = randn(d, d);
	
	CpuMatrix ac = randn(d, d);
	CpuMatrix bc = randn(d, d);
	
	for (int i = 0; i < 5; i++) {
		time_t now = clock();
		auto c = a.dot(b);
		auto cpu = c.cpu();
		double took = clock() - now;
		double millis = took / CLOCKS_PER_SEC * 1000.0;
		printf("millis: %8d\n", (int) millis);
	}
	
	println();
	for (int i = 0; i < 5; i++) {
		time_t now = clock();
		auto cc = ac.dot(bc);
		double took = clock() - now;
		double millis = took / CLOCKS_PER_SEC * 1000.0;
		printf("millis: %8d\n", (int) millis);
	}
}

void test1() {
	GpuMatrix a = { { 1, 2, 3 }, { 4, 5, 6 } };
	GpuMatrix b = { { 1, 2 }, { 3, 4 }, { 5, 6 } };
	
	auto scal = 2 * a;
	scal.print();
	
	auto d = a.dot(b);
	d.print();
	GpuVector v = { 1, 2 };
	auto c = a.affine(b, v);
	
	c.print();
}

void test2() {
	try {
		srand(time(NULL));
		
		Affine f = Affine();
		f.use_gpu(false);
		CpuMatrix x = { { 0 }, { 1 } };
		
		CpuMatrix y = { { 1 }, { 0 } };
		
		println("X:");
		x.print();
		
		println("Y");
		y.print();
		
		f.set_dim(x.n, y.n);
		f.init();
		CpuMatrix w = { { 0 } };
		CpuVector b = { 2 };
		f.set_weights(w);
		f.set_bias(b);
		
		float j;
		float alpha = 0.1;
		
		//f.print();
		int iter = 3;
		for (int i = 0; i <= iter; i++) {
			println("====================================");
			//f.print();
			Matrix& h = f.foward(x);
			if (iter < 10 || i % (iter / 10) == 0) {
				j = min_square_error(h, y);
				println("j: " + to_string(j));
			}
			
			h.print();
			f.print();
			CpuMatrix dg = cpu_cast(h) - y;
			
			f.backward(dg);
			
			f.update(alpha);
			
		}
		
		println("ended");
	} catch (Exception& ex) {
		println("Exception thrown");
		println(ex.what());
	}
}

void gpu_test() {
	try {
		srand(time(NULL));
		
		Affine f = Affine();
		f.use_gpu(true);
		GpuMatrix x = { { 0 }, { 1 } };
		
		GpuMatrix y = { { 1 }, { 0 } };
		
		println("X:");
		x.print();
		
		println("Y");
		y.print();
		
		f.set_dim(x.n, y.n);
		f.init();
		GpuMatrix w = { { 0 } };
		GpuVector b = { 2 };
		f.set_weights(w);
		f.set_bias(b);
		
		float j;
		float alpha = 0.1;
		
		//f.print();
		int iter = 100;
		for (int i = 0; i <= iter; i++) {
			//println("===================================");
			//f.print();
			Matrix& h = f.foward(x);
			
			if (iter <= 10 || i % (iter / 10) == 0) {
				j = min_square_error(h, y);
				//println("=======================================");
				println("j: " + to_string(j));
			}
			
			GpuMatrix dg = gpu_cast(h) - y;
			
			f.backward(dg);
			f.update(alpha);
		}
		
		Matrix& h = f.foward(x);
		
		h.print();
		println("ended");
	} catch (Exception& ex) {
		println("Exception thrown");
		println(ex.what());
	}
}

hipblasHandle_t cublas_handle = nullptr;

const char* _cuda_get_error_enum(hipblasStatus_t error) {
	switch (error) {
	case HIPBLAS_STATUS_SUCCESS:
		return "HIPBLAS_STATUS_SUCCESS";
		
	case HIPBLAS_STATUS_NOT_INITIALIZED:
		return "HIPBLAS_STATUS_NOT_INITIALIZED";
		
	case HIPBLAS_STATUS_ALLOC_FAILED:
		return "HIPBLAS_STATUS_ALLOC_FAILED";
		
	case HIPBLAS_STATUS_INVALID_VALUE:
		return "HIPBLAS_STATUS_INVALID_VALUE";
		
	case HIPBLAS_STATUS_ARCH_MISMATCH:
		return "HIPBLAS_STATUS_ARCH_MISMATCH";
		
	case HIPBLAS_STATUS_MAPPING_ERROR:
		return "HIPBLAS_STATUS_MAPPING_ERROR";
		
	case HIPBLAS_STATUS_EXECUTION_FAILED:
		return "HIPBLAS_STATUS_EXECUTION_FAILED";
		
	case HIPBLAS_STATUS_INTERNAL_ERROR:
		return "HIPBLAS_STATUS_INTERNAL_ERROR";
	}
	
	return "<unknown>";
}

void check_cublas(hipblasStatus_t status) {
	if (status == HIPBLAS_STATUS_SUCCESS) {
		return;
	}
	
	fprintf(stderr, "CUBLAS error %d\nMessage: %s.\n", status, _cuda_get_error_enum(status));
	fflush(stderr);
	throw Exception("Cuda error");
}

int main(void) {
	gpu_test();
	//test2();
	

	
	return 0;
}

