
#include <hip/hip_runtime.h>
/*
 * cuda_utils.cu
 *
 *  Created on: Feb 4, 2017
 *      Author: Yaison Alcantara
 */

#define BLOCK_SIZE_2D 16
#define BLOCK_SIZE_1D 256

namespace cs {
namespace gpu {

__global__ void cuda_kernel_matrix_mult(float* a, float* b, float* dest, unsigned int m, unsigned int n) {
	
	unsigned int i = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < m && j < n) {
		unsigned int absIdx = i * n + j;
		dest[absIdx] = a[absIdx] * b[absIdx];
	}
}

__global__ void cuda_kernel_vector_mult(float* a, float* b, float* dest, unsigned int l) {
	
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < l) {
		dest[idx] = a[idx] * b[idx];
	}
}

__global__ void cuda_kernel_vector_div(float* a, float* b, float* dest, unsigned int l) {
	
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < l) {
		dest[idx] = a[idx] / b[idx];
	}
}

__global__ void kernel_vector_pow(float* a, float exp, float* dest, unsigned int l) {
	
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < l) {
		dest[idx] = powf(a[idx], exp);
	}
}

__global__ void kernel_broadcast_sum_rows(float* a, float* b, float* dest, unsigned int m, unsigned int n) {
	
	unsigned int i = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < m && j < n) {
		unsigned int absIdx = i * n + j;
		dest[absIdx] = a[absIdx] + b[j];
	}
}

void cuda_matrix_mult(float* a, float* b, float* dest, size_t m, size_t n) {
	
	dim3 block(BLOCK_SIZE_2D, BLOCK_SIZE_2D);
	
	unsigned int blocksX = (unsigned int) ceil(n / (double) BLOCK_SIZE_2D);
	unsigned int blocksY = (unsigned int) ceil(m / (double) BLOCK_SIZE_2D);
	
	dim3 grid(blocksX, blocksY);
	
	cuda_kernel_matrix_mult<<<grid, block>>>(a, b, dest, m, n);
}

void cuda_vector_mult(float* a, float* b, float* dest, size_t length) {
	
	dim3 block(BLOCK_SIZE_1D);
	
	unsigned int blocksX = (unsigned int) ceil(length / (double) BLOCK_SIZE_1D);
	
	dim3 grid(blocksX);
	
	cuda_kernel_vector_mult<<<grid, block>>>(a, b, dest, length);
}

void cuda_vector_div(float* a, float* b, float* dest, size_t length) {
	
	dim3 block(BLOCK_SIZE_1D);
	
	unsigned int blocksX = (unsigned int) ceil(length / (double) BLOCK_SIZE_1D);
	
	dim3 grid(blocksX);
	
	cuda_kernel_vector_div<<<grid, block>>>(a, b, dest, length);
}

void cuda_vector_pow(float* a, float exp, float* dest, size_t l) {
	
	dim3 block(BLOCK_SIZE_1D);
	
	unsigned int blocksX = (unsigned int) ceil(l / (double) BLOCK_SIZE_1D);
	
	dim3 grid(blocksX);
	
	kernel_vector_pow<<<grid, block>>>(a, exp, dest, l);
}

void cuda_broadcast_sum_rows(float* a, float* b, float* dest, size_t m, size_t n) {
	
	dim3 block(BLOCK_SIZE_2D, BLOCK_SIZE_2D);
	
	unsigned int blocksX = (unsigned int) ceil(n / (double) BLOCK_SIZE_2D);
	unsigned int blocksY = (unsigned int) ceil(m / (double) BLOCK_SIZE_2D);
	
	dim3 grid(blocksX, blocksY);
	
	kernel_broadcast_sum_rows<<<grid, block>>>(a, b, dest, m, n);
}

}
 // namespace gpu
}// namespace cs
