
#include <hip/hip_runtime.h>
/*
 * cuda_utils.cu
 *
 *  Created on: Feb 4, 2017
 *      Author: Yaison Alcantara
 */

namespace cs {
namespace gpu {

unsigned int BLOCK_SIZE_2D = 16;
unsigned int BLOCK_SIZE_1D = 256;

__global__ void cuda_kernel_matrix_mult(float* a, float* b, float* dest, unsigned int m, unsigned int n) {
	
	unsigned int i = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < m && j < n) {
		unsigned int absIdx = i * n + j;
		dest[absIdx] = a[absIdx] * b[absIdx];
	}
}

__global__ void cuda_kernel_matrix_sum_rows(float* a, float* dest, unsigned int m, unsigned int n) {
	
	unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;
	if (j < n) {
		float sum = 0.0;
		for (unsigned int i = 0; i < m; i++) {
			sum += a[i * n + j];
		}
		dest[j] = sum;
	}
}

__global__ void cuda_kernel_vector_mult(float* a, float* b, float* dest, unsigned int l) {
	
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < l) {
		dest[idx] = a[idx] * b[idx];
	}
}

__global__ void cuda_kernel_vector_div(float* a, float* b, float* dest, unsigned int l) {
	
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < l) {
		dest[idx] = a[idx] / b[idx];
	}
}

__global__ void kernel_vector_pow(float* a, float exp, float* dest, unsigned int l) {
	
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < l) {
		dest[idx] = powf(a[idx], exp);
	}
}

__global__ void kernel_broadcast_sum_rows(float* a, float* b, float* dest, unsigned int m, unsigned int n) {
	
	unsigned int i = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < m && j < n) {
		unsigned int absIdx = i * n + j;
		dest[absIdx] = a[absIdx] + b[j];
	}
}

void cuda_matrix_mult(float* a, float* b, float* dest, size_t m, size_t n) {
	
	dim3 block(BLOCK_SIZE_2D, BLOCK_SIZE_2D);
	
	unsigned int blocksX = (unsigned int) ceil(n / (double) BLOCK_SIZE_2D);
	unsigned int blocksY = (unsigned int) ceil(m / (double) BLOCK_SIZE_2D);
	
	dim3 grid(blocksX, blocksY);
	
	cuda_kernel_matrix_mult<<<grid, block>>>(a, b, dest, m, n);
}

void cuda_vector_mult(float* a, float* b, float* dest, size_t length) {
	
	dim3 block(BLOCK_SIZE_1D);
	
	unsigned int blocksX = (unsigned int) ceil(length / (double) BLOCK_SIZE_1D);
	
	dim3 grid(blocksX);
	
	cuda_kernel_vector_mult<<<grid, block>>>(a, b, dest, length);
}

void cuda_vector_div(float* a, float* b, float* dest, size_t length) {
	
	dim3 block(BLOCK_SIZE_1D);
	
	unsigned int blocksX = (unsigned int) ceil(length / (double) BLOCK_SIZE_1D);
	
	dim3 grid(blocksX);
	
	cuda_kernel_vector_div<<<grid, block>>>(a, b, dest, length);
}

void cuda_vector_pow(float* a, float exponent, float* dest, size_t l) {
	
	dim3 block(BLOCK_SIZE_1D);
	
	unsigned int blocksX = (unsigned int) ceil(l / (double) BLOCK_SIZE_1D);
	
	dim3 grid(blocksX);
	
	kernel_vector_pow<<<grid, block>>>(a, exponent, dest, l);
}

void cuda_broadcast_sum_rows(float* a, float* b, float* dest, size_t m, size_t n) {
	
	dim3 block(BLOCK_SIZE_2D, BLOCK_SIZE_2D);
	
	unsigned int blocksX = (unsigned int) ceil(n / (double) BLOCK_SIZE_2D);
	unsigned int blocksY = (unsigned int) ceil(m / (double) BLOCK_SIZE_2D);
	
	dim3 grid(blocksX, blocksY);
	
	kernel_broadcast_sum_rows<<<grid, block>>>(a, b, dest, m, n);
}

void cuda_sum_rows(float* a, float* dest, size_t m, size_t n) {
	
	dim3 block(BLOCK_SIZE_1D);
	//Note: in this case is n as the number of threads to use, cuz each thread is doing a reduce operation on the rows
	unsigned int blocksX = (unsigned int) ceil(n / (double) BLOCK_SIZE_1D);
	
	dim3 grid(blocksX);
	
	cuda_kernel_matrix_sum_rows<<<grid, block>>>(a, dest, m, n);
}

}
 // namespace gpu
}// namespace cs
